#include <stdbool.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>


#define MAX_THREADS 1024

__global__ void bitonicSortGpuStep(int *deviceValues, int j, int k) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x, ixj = i ^ j;

    if (ixj > i) {
        if ((i&k) == 0) {
            if (deviceValues[i] > deviceValues[ixj]) {
                int temp = deviceValues[i];
                deviceValues[i] = deviceValues[ixj];
                deviceValues[ixj] = temp;
            }
        } else {
            if (deviceValues[i] < deviceValues[ixj]) {
                int temp = deviceValues[i];
                deviceValues[i] = deviceValues[ixj];
                deviceValues[ixj] = temp;
            }
        }
    }
}

extern "C"
void bitonicSortGpu(int *hostValues, int numVals)
{
    int *deviceValues;
    size_t size = numVals * sizeof(int);

    hipError_t hipError_t = hipMalloc((void**) &deviceValues, size);
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "Cannot allocate Gpu memory for deviceValues: %s\n",
                hipGetErrorString(hipError_t));
        exit(1);
    }

    hipError_t = hipMemcpy(deviceValues, hostValues, size, hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "Cannot copy data from hostValues to deviceValues: %s\n",
                hipGetErrorString(hipError_t));
        exit(1);
    }

    int numThreads;
    int numBlocks;

    if (numVals <= MAX_THREADS) {
        numThreads = numVals;
        numBlocks = 1;
    } else {
        numThreads = MAX_THREADS;
        numBlocks = numVals / numThreads;
    }

    dim3 blocks(numBlocks, 1);
    dim3 threads(numThreads, 1);

    for (int k = 2; k <= numVals; k <<= 1) {
        for (int j = k >> 1; j > 0; j >>= 1) {
            bitonicSortGpuStep<<<blocks, threads>>>(deviceValues, j, k);

            hipError_t = hipGetLastError();
            if (hipError_t != hipSuccess) {
                fprintf(stderr, "Cannot launch CUDA kernel: %s\n",
                        hipGetErrorString(hipError_t));
                exit(1);
            }
        }
    }

    hipError_t = hipMemcpy(hostValues, deviceValues, size, hipMemcpyDeviceToHost);
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "Cannot copy data from deviceValues to hostValues: %s\n",
                hipGetErrorString(hipError_t));
        exit(1);
    }

    hipError_t = hipFree(deviceValues);
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "Cannot free Gpu memory for deviceValues: %s\n",
                hipGetErrorString(hipError_t));
        exit(1);
    }
}